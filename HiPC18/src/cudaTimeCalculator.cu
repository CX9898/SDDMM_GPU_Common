#include "cudaTimeCalculator.cuh"
#include "cudaErrorCheck.cuh"

cudaTimeCalculator::cudaTimeCalculator(){
    _time = 0.0f;

    cudaErrCheck(hipEventCreate(&_star));
    cudaErrCheck(hipEventCreate(&_stop));
}

cudaTimeCalculator::~cudaTimeCalculator() {
    cudaErrCheck(hipEventDestroy(_star));
    cudaErrCheck(hipEventDestroy(_stop));
}

void cudaTimeCalculator::startClock() {
    cudaErrCheck(hipEventRecord(_star));
}

void cudaTimeCalculator::endClock() {
    cudaErrCheck(hipEventRecord(_stop));
    cudaErrCheck(hipEventSynchronize(_stop));
}

float cudaTimeCalculator::getTime() {
    cudaErrCheck(hipEventElapsedTime(&_time, _star, _stop));
    return _time;
}