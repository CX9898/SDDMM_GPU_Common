#include "CudaTimeCalculator.cuh"
#include "cudaErrorCheck.cuh"

CudaTimeCalculator::CudaTimeCalculator(){
    _time = 0.0f;

    cudaErrCheck(hipEventCreate(&_star));
    cudaErrCheck(hipEventCreate(&_stop));
}

CudaTimeCalculator::~CudaTimeCalculator() {
    cudaErrCheck(hipEventDestroy(_star));
    cudaErrCheck(hipEventDestroy(_stop));
}

void CudaTimeCalculator::startClock() {
    cudaErrCheck(hipEventRecord(_star));
}

void CudaTimeCalculator::endClock() {
    cudaErrCheck(hipEventRecord(_stop));
    cudaErrCheck(hipEventSynchronize(_stop));
}

float CudaTimeCalculator::getTime() {
    cudaErrCheck(hipEventElapsedTime(&_time, _star, _stop));
    return _time;
}